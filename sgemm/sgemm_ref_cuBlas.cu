#include "hip/hip_runtime.h"
#include "params.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define CEIL_DIV(x,y) ((x + y - 1) / y)
// #define ENABLE_CPU_GEMM

void cpu_gemm(const float *A, const float *B, float *C, int M, int N, int K) {
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            C[i * N + j] = 0;
            for (int k = 0; k < K; k++) {
                C[i * N + j] += A[i * K + k] * B[k * N + j];
            }
        }
    }
}

int main() {
    // int M = 2048;
    // int N = 2048;
    // int K = 2048;

    const size_t sizeA = M * K * sizeof(float);
    const size_t sizeB = K * N * sizeof(float);
    const size_t sizeC = M * N * sizeof(float);

    // Allocate memory on the host
    float *A = (float *)malloc(sizeA);
    float *B = (float *)malloc(sizeB);

    float *C_cpu = (float *)malloc(sizeC);
    float *C_gpu = (float *)malloc(sizeC);

    if(A == NULL || B == NULL || C_cpu == NULL || C_gpu == NULL) {
        fprintf(stderr, "Failed to allocate host memory\n");
        return -1;
    }

    // Initialize matrices A and B
    srand(time(NULL));
    
    // Initialize A with random values between -1 and 1
    for (int i = 0; i < M * K; i++) {
        A[i] = 2.0f * rand() / RAND_MAX - 1.0f;
    }
    
    // Initialize B with random values between -1 and 1
    for (int i = 0; i < K * N; i++) {
        B[i] = 2.0f * rand() / RAND_MAX - 1.0f;
    }

    // Allocate memory on the device
    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, sizeA);
    hipMalloc((void **)&d_B, sizeB);
    hipMalloc((void **)&d_C, sizeC);
    if (d_A == NULL || d_B == NULL || d_C == NULL) {
        fprintf(stderr, "Failed to allocate device memory\n");
        return -1;
    }

    // Copy matrices A and B from host to device
    hipMemcpy(d_A, A, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, sizeB, hipMemcpyHostToDevice);

    // Launch the kernel
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Init cuBlas
    hipblasHandle_t handle;
    hipblasStatus_t status = hipblasCreate(&handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "Failed to create cuBLAS handle\n");
        return -1;
    }

    float alpha = 1.0f;
    float beta = 0.0f;
    for(int i = 0; i < 3; i++) {
        hipEventRecord(start);
        status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K,
                    &alpha, d_A, M, d_B, K, &beta, d_C, M);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time taken for GPU's SGEMM: %f ms\n", milliseconds);
    
    // Calculate throughput in GFLOPS
    double seconds = milliseconds / 1000.0;
    double operations = 2.0 * M * N * K;  // Multiply-add operations
    double throughput = operations / seconds / 1e9;
    printf("Throughput: %.2f GFLOPS\n", throughput);
    
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Get the result back to the host
    hipMemcpy(C_gpu, d_C, sizeC, hipMemcpyDeviceToHost);

#ifdef ENABLE_CPU_GEMM
    // Perform CPU computation for verification
    clock_t start_cpu, end_cpu;
    start_cpu = clock();
    cpu_gemm(A, B, C_cpu, M, N, K);
    end_cpu = clock();

    double cpu_time = double(end_cpu - start_cpu) / CLOCKS_PER_SEC;
    printf("Time taken for CPU's SGEMM: %f seconds\n", cpu_time);
    
    // Calculate CPU throughput in GFLOPS
    // double operations = 2.0 * M * N * K;  // Multiply-add operations
    double cpu_throughput = operations / cpu_time / 1e9;
    printf("CPU Throughput: %.2f GFLOPS\n", cpu_throughput);


    // Verify the result
    for (int i = 0; i < M * N; i++) {
        if (fabs(C_cpu[i] - C_gpu[i]) > 1e-3) {
            fprintf(stderr, "Mismatch at index %d: CPU = %f, GPU = %f\n", i, C_cpu[i], C_gpu[i]);
            break;
        }
    }
#endif
    printf("SGEMM computation completed successfully.\n");

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    // Free host memory
    free(A);
    free(B);

    free(C_cpu);
    free(C_gpu);

    hipblasDestroy(handle);

    // Reset the device and exit
    hipDeviceReset();
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
        return -1;
    }

    return 0;

}